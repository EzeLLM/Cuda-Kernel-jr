#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void cuda_hello(){
    printf("Hello from the gpu brah");
}

int main(){
    """
    the statement below tells cuda to run this function
    with 1 block and one thread, im still unclear on this
    side. gotta learn.
    """
    cuda_hello<<<1,1>>>();

    """
    The gpu may be still running even though
    the cpu has finished the host code. That is
    the 'why' of me not being able to see the print
    statement without sychronizing cuda with the cpu
    """
    hipDeviceSynchronize();


    """
    The code below is a 'kinda convention' to add
    at the end of cuda programs to reset the device state
    """
    hipDeviceReset();  
    
    return 0;
}