#include "hip/hip_runtime.h"
#include <stdio.h>
#define Stuff 100
__global__ void va(float *out, float *a, float *b,int n){
    for (int i=0;i<n;i++){
        out[i] = a[i] + b[i];
    }
}



int main(){
    """declaring pointers"""
    float *a,*b,*out;
    float *d_a,*d_b,*d_out; // this is a pointer allocated on gpu
                // we use d_ prefix to indicate that
                // the prefix is just a convention
    """we allocate memory on cpu(host) for the vectors"""
    a = (float*)malloc(sizeof(float)*Stuff)
    b = (float*)malloc(sizeof(float)*Stuff)
    c = (float*)malloc(sizeof(float)*Stuff)
    // Now we allocate memory on the gpu(device)
    // Cuda malloc works differently, we pass pointer
    // then cuda modifies it and allocate if the size we
    // passed. cuda malloc returns an error code instead of pointer.
    // im don't have the expertise required to simplify the (void**)&d_a part
    // but here it is: &d_a is the address. cuda expects void address, 
    // so we cast d_a to (void**). we use ** to be passing
    // by refrence not by value. so at the end we
    // pass the pointer of d_a (not a value of the pointer)
    // casted to void.
    hipMalloc((void**)&d_a, sizeof(float)*Stuff)
    hipMalloc((void**)&d_b, sizeof(float)*Stuff)
    hipMalloc((void**)&d_out, sizeof(float)*Stuff)
    // You can initialize the vectors with a for loop now
    // then move them to gpu memory allocated by 
    // hipMemcpy(void *d_a, void *a, size_t sizeof(float)*Stuff, hipMemcpyKind hipMemcpyHostToDevice)
    // But i want to write a function to initialize two vectors in parallel.

    // Im leaving to study now, will implement this later as it needs
    // more study, as it is more complex. 
    // **Commit**

}